#include <iostream>
#include <hip/hip_runtime.h>

// Helper macro to check for CUDA errors
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess) 
    {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

// CUDA kernel: 1 thread computes the Fibonacci sequence
__global__ void fibonacci_kernel(unsigned long long* fib, int N) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        fib[0] = 0;
        if (N > 1) fib[1] = 1;

        for (int i = 2; i < N; i++) {
            fib[i] = fib[i-1] + fib[i-2];
        }
    }
}

// Sequential CPU version
void fibonacci_seq(unsigned long long* fib, int N) {
    fib[0] = 0;
    if (N > 1) fib[1] = 1;
    for (int i = 2; i < N; ++i) {
        fib[i] = fib[i - 1] + fib[i - 2];
    }
}

int main() {
    const int N = 220;
    unsigned long long *h_fib_seq = new unsigned long long[N];
    unsigned long long *h_fib_gpu = new unsigned long long[N];

    // CPU computation
    fibonacci_seq(h_fib_seq, N);

    // Allocate memory on GPU
    unsigned long long* d_fib;
    gpuErrchk(hipMalloc((void**)&d_fib, N * sizeof(unsigned long long)));

    // 🔥 Important: Initialize device memory to 0
    gpuErrchk(hipMemset(d_fib, 0, N * sizeof(unsigned long long)));

    // Launch kernel
    fibonacci_kernel<<<1, 1>>>(d_fib, N);

    // Check for errors
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());

    // Copy result back to host
    gpuErrchk(hipMemcpy(h_fib_gpu, d_fib, N * sizeof(unsigned long long), hipMemcpyDeviceToHost));

    // Compare CPU and GPU results
    bool correct = true;
    for (int i = 0; i < N; ++i) {
        if (h_fib_seq[i] != h_fib_gpu[i]) {
            correct = false;
            std::cout << "Mismatch at index " << i << ": CPU " << h_fib_seq[i] << " GPU " << h_fib_gpu[i] << std::endl;
            break;
        }
    }
    if (correct) {
        std::cout << "✅ CPU and GPU Fibonacci sequences match!" << std::endl;
    }

    // Print first 20 Fibonacci numbers
    for (int i = 0; i < 20; ++i) {
        std::cout << "F(" << i << ") = " << h_fib_gpu[i] << std::endl;
    }

    // Free memory
    hipFree(d_fib);
    delete[] h_fib_seq;
    delete[] h_fib_gpu;

    return 0;
}
